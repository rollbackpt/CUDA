#include "hip/hip_runtime.h"
/*
Based on the hello-world created by Ingemar Ragnemalm 2010
(http://computer-graphics.se/hello-world-for-cuda.html)
and the book "CUDA by Example"

This example code detects CUDA devices, print their information
and tests the parallel programing using CUDA

Author: João Ribeiro

nvcc check-cuda.cu -L /usr/local/cuda/lib -lcudart -o check-cuda
*/

#include <stdio.h>
#include <unistd.h>

const int N = 16;
const int blocksize = 16;

__global__
void hello(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

int main()
{
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

	char *ad;
	int *bd;
	int dev_count;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);
	hipDeviceProp_t prop;

	hipGetDeviceCount(&dev_count);
	printf("Number of CUDA devices found: %d\n\n", dev_count);

	/* Get and print GPU information */
	for (int i = 0; i < dev_count; i++) {
		hipGetDeviceProperties(&prop, i);

		printf( "--- General Information for device %d ---\n", i );
		printf( "Name: %s\n", prop.name );
		printf( "Compute capability: %d.%d\n", prop.major, prop.minor );
		printf( "Clock rate: %d\n", prop.clockRate );
		printf( "Device copy overlap:" );

		if (prop.deviceOverlap)
			printf( "Enabled\n" );
		else
			printf( "Disabled\n" );

		printf( "Kernel execition timeout :" );

		if (prop.kernelExecTimeoutEnabled)
			printf( "Enabled\n" );
		else
			printf( "Disabled\n" );

		printf( "--- Memory Information for device %d ---\n", i );
		printf( "Total global mem: %ld\n", prop.totalGlobalMem );
		printf( "Total constant Mem: %ld\n", prop.totalConstMem );
		printf( "Max mem pitch: %ld\n", prop.memPitch );
		printf( "Texture Alignment: %ld\n", prop.textureAlignment );
		printf( "--- MP Information for device %d ---\n", i );
		printf( "Multiprocessor count: %d\n",prop.multiProcessorCount );
		printf( "Shared mem per mp: %ld\n", prop.sharedMemPerBlock );
		printf( "Registers per mp: %d\n", prop.regsPerBlock );
		printf( "Threads in warp: %d\n", prop.warpSize );
		printf( "Max threads per block: %d\n", prop.maxThreadsPerBlock );
		printf( "Max thread dimensions: (%d, %d, %d)\n",
		prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
		printf( "Max grid dimensions:(%d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2] );
		printf( "\n" );
	}
	/* End of print GPU information */

	printf("The next print will be the result of a parallel processed array. If you see the string \"Hello World!\" then CUDA is working!\n\n");
	printf("%s", a);

	/* Using CUDA to generate the string "World!"*/
	hipMalloc( (void**)&ad, csize );
	hipMalloc( (void**)&bd, isize );
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
	hipFree( ad );
	hipFree( bd );
	/* End of using CUDA to generate the string "World!"*/

	printf("%s\n\n", a);
	usleep(1000);
	return EXIT_SUCCESS;
}
